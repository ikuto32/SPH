#include "hip/hip_runtime.h"
#include "sph/gpu/hash_grid_2d.hpp"
#include <hip/hip_runtime.h>
#include <vector>
#include "../src/sph/debug_gpu.hpp"
#include <cassert>

using namespace sph;

int main() {
#ifdef SPH_ENABLE_HASH2D
    const uint32_t Nx = 32;
    const uint32_t Ny = 32;
    const uint32_t N = Nx * Ny;

    std::vector<float2> hPos(N);
    for (uint32_t i = 0; i < N; ++i) {
        hPos[i] = make_float2(static_cast<float>(i % Nx), static_cast<float>(i / Nx));
    }

    ParticleSoA p{};
    CUDA_TRY(hipMalloc(&p.pos, N * sizeof(float2)));
    CUDA_TRY(hipMemcpy(p.pos, hPos.data(), N * sizeof(float2), hipMemcpyHostToDevice));

    HashGrid2D grid{};
    grid.gridDim = make_uint2(Nx, Ny);
    grid.invCell = 1.0f;
    grid.gridCells = Nx * Ny;
    grid.particles = p;
    CUDA_TRY(hipMallocManaged(&grid.hashBuf, N * sizeof(uint32_t)));
    CUDA_TRY(hipMallocManaged(&grid.idxBuf, N * sizeof(uint32_t)));
    CUDA_TRY(hipMallocManaged(&grid.cellStart, grid.gridCells * sizeof(uint32_t)));
    CUDA_TRY(hipMallocManaged(&grid.cellEnd, grid.gridCells * sizeof(uint32_t)));

    grid.build(N, 0);
    CUDA_TRY(hipDeviceSynchronize());

    uint32_t* neigh;
    uint32_t* count;
    CUDA_TRY(hipMallocManaged(&neigh, N * MAX_NEIGHBORS * sizeof(uint32_t)));
    CUDA_TRY(hipMallocManaged(&count, N * sizeof(uint32_t)));

    float hh = 1.1f;
    grid.findNeighbors(N, hh, neigh, count);
    CUDA_TRY(hipDeviceSynchronize());

    std::vector<uint32_t> cpuCount(N, 0);
    float hh2 = hh * hh;
    for (uint32_t i = 0; i < N; ++i) {
        float2 pi = hPos[i];
        for (uint32_t j = 0; j < N && cpuCount[i] < MAX_NEIGHBORS; ++j) {
            float2 pj = hPos[j];
            float dx = pi.x - pj.x;
            float dy = pi.y - pj.y;
            if (dx * dx + dy * dy <= hh2) {
                cpuCount[i]++;
            }
        }
    }

    for (uint32_t i = 0; i < N; ++i) {
        assert(count[i] == cpuCount[i]);
    }

    CUDA_TRY(hipFree(neigh));
    CUDA_TRY(hipFree(count));
    CUDA_TRY(hipFree(p.pos));
    CUDA_TRY(hipFree(grid.hashBuf));
    CUDA_TRY(hipFree(grid.idxBuf));
    CUDA_TRY(hipFree(grid.cellStart));
    CUDA_TRY(hipFree(grid.cellEnd));
#endif
    return 0;
}
