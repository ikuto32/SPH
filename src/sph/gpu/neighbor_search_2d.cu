#include "hip/hip_runtime.h"
#include "hash_grid_2d.hpp"
#include <hip/hip_runtime.h>

namespace sph {

#ifndef MAX_NEIGHBORS
#define MAX_NEIGHBORS 64
#endif

__global__
void neighbourSearchKernel(const float2* pos,
                           const uint32_t* cellStart,
                           const uint32_t* cellEnd,
                           const uint32_t* idxBuf,
                           float hh,
                           uint2 gridDim,
                           float invCell,
                           uint32_t N,
                           uint32_t* outCount) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    // Placeholder: zero neighbours
    outCount[i] = 0;
}

} // namespace sph
