#include "hip/hip_runtime.h"
#include "kernels_cuda.h"

#ifdef USE_CUDA

#include <cmath>          // 必須
#ifndef M_PI              // まだ無ければ自前で定義
#define M_PI 3.14159265358979323846
#endif

namespace sph {

__global__ void calcSmoothingKernelKernel(const float* dist, float* out, float radius, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float d = dist[idx];
        float val = 0.0f;
        if (d < radius) {
            float volume = (float)(M_PI * radius * radius * radius * radius) / 6.0f;
            float t = radius - d;
            val = t * t / volume;
        }
        out[idx] = val;
    }
}

void calcSmoothingKernelCUDA(const float* dist, float* out, float radius, int n)
{
    float* d_in = nullptr;
    float* d_out = nullptr;
    CUDA_CHECK(hipMalloc(&d_in, n * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_out, n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_in, dist, n * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    calcSmoothingKernelKernel<<<blocks, threads>>>(d_in, d_out, radius, n);
    CUDA_KERNEL_CHECK();
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipMemcpy(out, d_out, n * sizeof(float), hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_in));
    CUDA_CHECK(hipFree(d_out));
}

} // namespace sph

#endif // USE_CUDA
